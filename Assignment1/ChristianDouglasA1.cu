//Christian Douglas

#include <hip/hip_runtime.h>
#include <stdio.h>

const int arrSize = 4096;          //max size desired for each array used
const int blockSize = 1024;        //largest number of threads in a block

//kernal function to get the value for each thread allocated to pointers
//a, b, and c and adds the values from a and b and assigns them to c
__global__
void arraySum(int *a, int *b, int *c)
{ 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    c[i] = a[i] + b[i]; 
}

int main()
{
    int A[arrSize], B[arrSize], C[arrSize]; //
    int *a_d, *b_d, *c_d;

    //Initializes each element of A from 0 - 4095 
    //and B from 4095 to 8191
    for(int i = 0; i < arrSize; i++)
    {
        A[i] = i;
        B[i] = (arrSize-1)+i;
    }

    
    //allocating pointers to size 4096
    hipMalloc((void**)&a_d, arrSize);
    hipMalloc((void**)&b_d, arrSize);
    hipMalloc((void**)&c_d, arrSize);

    //moves values from A, B, C to their respective pointers and moves the 
    //values to the gpu
    hipMemcpy(a_d, A, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(c_d, C, arrSize, hipMemcpyHostToDevice);

    //grid is set to 12 blocks and block is set to 1024 threads
    dim3 dimGrid(12, 1);
    dim3 dimBlock(blockSize, 1);

    //calls the function arraySum to add the values allocated in the pointers
    arraySum<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);

    //the values from the c pointer are moved back to C array and moved back 
    //to the cpu
    hipMemcpy(C, c_d, arrSize, hipMemcpyDeviceToHost);

    //deallocates the pointers to prevent memory leaks
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
    //Prints the first and last values of the C array
    //The values should be 4095 and 12186
    printf("%s", "Array C's first element: ");
    printf("%d", C[0]);
    printf("%c", '\n');
    printf("%s", "Last Element: ");
    printf("%d", C[arrSize-1]);
    printf("%c", '\n');

    return 0;
}
#include "hip/hip_runtime.h"
//Christian Douglas
#include <stdio.h>

const int arrSize = 4096;          //max size desired for each array used
const int blockSize = 1024;        //largest number of threads in a block

__global__
void allocArrays(float *A, float *B, float *C)
{
    float *a_d, *b_d, *c_d;
    int size = arrSize * sizeof(float);

    //allocating pointers to size 4096
    hipMalloc((void**)&a_d, size);
    hipMalloc((void**)&b_d, size);
    hipMalloc((void**)&c_d, size);

    //moves values from A, B to their respective pointers and moves the 
    //values to the gpu
    hipMemcpy(a_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, size, hipMemcpyHostToDevice);

    //grid is set to 4 blocks and block is set to 1024 threads
    dim3 dimGrid(4, 1);
    dim3 dimBlock(blockSize, 1);

    //calls the function arraySum to add the values allocated in the pointers
    arraySum<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, size);

    //the values from the c pointer are moved back to C array and moved back 
    //to the cpu
    hipMemcpy(C, c_d, size, hipMemcpyDeviceToHost);

    //deallocates the pointers to prevent memory leaks
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

//kernal function to get the value for each thread allocated to pointers
//a, b, and c and adds the values from a and b and assigns them to c
__device__
void arraySum(float *a, float *b, float *c, float size)
{ 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size) c[i] = a[i] + b[i]; 
}

int main()
{
    float A[arrSize], B[arrSize], C[arrSize]; 
    
    //Initializes each element of A from 0 - 4095 
    //and B from 4095 to 8191
    for(int i = 0; i < arrSize; i++)
    {
        A[i] = i;
        B[i] = (arrSize-1)+i;
    }

    allocArrays(A, B, C);

    //Prints the first and last values of the C array
    //The values should be 4095 and 12186
    printf("%s", "Array C's first element: ");
    printf("%d", C[0]);
    printf("%c", '\n');
    printf("%s", "Last Element: ");
    printf("%d", C[arrSize-1]);
    printf("%c", '\n');

    return 0;
}
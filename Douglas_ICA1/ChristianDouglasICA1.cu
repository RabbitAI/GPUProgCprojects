#include "hip/hip_runtime.h"
//Christian Douglas
#include <stdio.h>

const int N = 10240;
const int numThread = 1024;

__global__
void arrProduct(int* a, int* b, int* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    c[i] = (a[i]*(i<N))*(b[i]*(i<N));

}

int main()
{
    int A[N], B[N], C[N];
    int *a_d, *b_d, *c_d;
    int index = 0, twoblock = 2, tenblock = 10;
    while(index != 2)
    {
        for(int i = 0; i < N; i++)
        {
            A[i] = 2 * i;
            B[i] = (2*i)+1;
        }

        hipMalloc((void**)&a_d, N);
        hipMalloc((void**)&b_d, N);
        hipMalloc((void**)&c_d, N);
        hipMemcpy(a_d, A, N, hipMemcpyHostToDevice);
        hipMemcpy(b_d, B, N, hipMemcpyHostToDevice);
        hipMemcpy(c_d, C, N, hipMemcpyHostToDevice);

        if(index != 2) {dim3 dimGrid(twoblock, 1);}
        else {dim3 dimGrid(tenblock,1);}

        dim3 dimBlock(numThread,1);

        arrProduct<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);

        hipMemcpy(C, c_d, N, hipMemcpyDeviceToHost);

        hipFree(a_d);
        hipFree(b_d);
        hipFree(c_d);
        
        if(index != 2) { printf("%d", twoblock);}
        else { printf("%d", tenblock);}
        printf("%s Blocks (C[0], C[");
        printf("%d",N);
        printf("%s] = (");
        printf("%d", C[0]);
        printf("%s, ");
        printf("%d", C[N-1]);
        printf("%s)\n");


        index++;
    }
    return 0;
}
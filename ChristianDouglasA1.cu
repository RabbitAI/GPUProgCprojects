#include "hip/hip_runtime.h"
//Christian Douglas
#include <stdio.h>

const int arrSize = 4096;
const int blockSize = 1024;

__global__
void arrayProduct()
{
    

    

}

int main()
{
    const THREADSIZE = 1023;
    int A[arrSize], B[arrSize], C[arrSize];
    
    for(int i = 0; i < arrSize; i++)
    {
        A[i] = i;
        B[i] = (arrSize-1)+i;
    }

    for(int i = 0; i< 4; i++)
    {
        hipMalloc((void**)&a_d, blockSize);
        hipMalloc((void**)&b_d, blockSize);
        hipMalloc((void**)&c_d, blockSize);

        hipMemcpy(a_d, A, THREADSIZE, hipMemcpyHostToDevice);
        hipMemcpy(b_d, B, THREADSIZE, hipMemcpyHostToDevice);
        hipMemcpy(c_d, C, THREADSIZE, hipMemcpyHostToDevice);
    }
    // printf("%d", A[0]);
    // printf("%c", ' ');
    // printf("%d", A[arrSize-1]);
    // printf("%c", '\n');
    // printf("%d", B[0]);
    // printf("%c", ' ');
    // printf("%d", B[arrSize-1]);

    return 0;
}
#include "hip/hip_runtime.h"
//Christian Douglas
#include <stdio.h>

const int arrSize = 4096;
const int blockSize = 1024;

__global__
void arrayProduct(int *a, int *b, int *c)
{
    
    c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x]; 
    

}

int main()
{
    int A[arrSize], B[arrSize], C[arrSize];
    dim3 dimGrid(3, 1);
    dim3 dimBlock(blockSize, 1);

    for(int i = 0; i < arrSize; i++)
    {
        A[i] = i;
        B[i] = (arrSize-1)+i;
    }

    for(int i = 0; i< arrSize; i*blockSize)
    {
        hipMalloc((void**)&a_d, blockSize);
        hipMalloc((void**)&b_d, blockSize);
        hipMalloc((void**)&c_d, blockSize);

        hipMemcpy(a_d, A, blockSize, hipMemcpyHostToDevice);
        hipMemcpy(b_d, B, blockSize, hipMemcpyHostToDevice);
        hipMemcpy(c_d, C, blockSize, hipMemcpyHostToDevice);
        arrayProduct<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);

        hipMemcpy(C, c_d, blockSize, hipMemcpyDeviceToHost);
        hipFree(a_d);
        hipFree(b_d);
        hipFree(c_d);
    }

    // printf("%d", A[0]);
    // printf("%c", ' ');
    // printf("%d", A[arrSize-1]);
    // printf("%c", '\n');
    // printf("%d", B[0]);
    // printf("%c", ' ');
    // printf("%d", B[arrSize-1]);

    return 0;
}
#include "hip/hip_runtime.h"
//Christian Douglas
#include <stdio.h>

const int arrSize = 4096;
const int blockSize = 1024;

__global__
void arraySum(int *a, int *b, int *c)
{
    i = blockIdx.x*dimBlock.x+threadIdx.x;

    c[i] = a[i] + b[i]; 
}

int main()
{
    int A[arrSize], B[arrSize], C[arrSize];
    int *a_d, *b_d, *c_d;
    dim3 dimGrid(4, 1);
    dim3 dimBlock(blockSize, 1);

    for(int i = 0; i < arrSize; i++)
    {
        A[i] = i;
        B[i] = (arrSize-1)+i;
    }


    hipMalloc((void**)&a_d, arrSize);
    hipMalloc((void**)&b_d, arrSize);
    hipMalloc((void**)&c_d, arrSize);

    hipMemcpy(a_d, A, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(c_d, C, arrSize, hipMemcpyHostToDevice);
    arraySum<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);

    hipMemcpy(C, c_d, arrSize, hipMemcpyDeviceToHost);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    

    printf("%s", "Array C's first element: ");
    printf("%d", C[0]);
    printf("%c", '\n');
    printf("%s", "Last Element: ");
    printf("%d", C[arrSize-1]);
    

    return 0;
}
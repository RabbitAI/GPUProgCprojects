#include "hip/hip_runtime.h"
//**************************************************************
// Assignment ChristianDouglasA3
// Name: Chistian Douglas
// GPU Programming Date: 10/14/2020
//*******************************************************************
//Program finds the product of 2 arrays A and B, and assigns the 
//values to array C. Each value of array A is initialized by the
//equation 2*i and each value of Array B is initialized by the 
//equation 2*i+1. All 3 arrays are set to the size of 10240 
//elements by using the global constant N. The main function also
//contains 2 constant integers blockTwo, which is assigned the 
//value of 2 and blockTen, which is assigned the value of 10,
//these two constants are used to let the gpu know how many blocks
//to use. A integer variable called size is also in main and is 
//equal to the value of N * the size of the data type float and is 
//used when allocating space on the cpu for the pointers a_d, b_d,
//and c_d in the funtion allocArray using hipMalloc. The pointers
//a_d, b_d, c_d point to float data types, after being allocated
//to the size of size, the values of arrays A and B are then copied
//to a_d and b_d using hipMemcpy. dimGrid's number of blocks is 
//defined by the variable block and dimBlock's number of threads
//is set t0 1024, which is the max number of threads a block can 
//hold. The kernel of arrayProduct is then called, which then takes
//the total number of threads multiplied by the total number of 
//blocks added to the thread ID's and performs the multiplication 
//of the elements assigned to pointers a_d and b_d, assigning them
//to c_d's elements. The values in c_d are the copied to the elements
//of array C and the the memory allocated to the pointers of a_d,
//b_d, and c_d are freed using hipFree. The function printVal prints
//the first and last value of array C. 
//*******************************************************************
#include <stdio.h>

const int N = 10240;            //number of elements in arrays

//*******************************************************************
//Method Name: allocArray
//Parameters: A, B, C, size, block
//Purpose: Allocates size of size to the local pointers a_d, b_d, and
//c_d and copies the values from the arrays A, B, and C using 
//cudaMemcyp. dimGrid size is defind by the integer block and 
//dimBlock is defined by the max number of threads, 1024, 
//arrayProduct is called, after c_d's values are copied to array C
//and the pointers are freed using hipFree
//*******************************************************************
void allocArray(float *A, float *B, float *C, int size, int block)
{
    float *a_d, *b_d, *c_d;

    hipMalloc((void**)a_d, size);
    hipMalloc((void**)b_d, size);
    hipMalloc((void**)c_d, size);

    hipMemcpy(a_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, size, hipMemcpyHostToDevice);
    
    dim3 dimGrid(block, 1);
    dim3 dimBlock(1024, 1);

    arrayProduct<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, size);

    hipMemcpy(C, c_d, size, hipMemcpyDeviceToHost);
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

//*******************************************************************
//Kernel Name: arrayProduct
//Parameters: a_d, b_d, c_d, size
//Purpose: Integer i is defined by the total number of threads
//multiplied by the total number of block added to the threadID 
//number. The pointers then use the value of i to access the values
//they contain and uses parallel programming to obtain all the 
//product values of pointers a_d and b_d and assigns them to c_d.
//*******************************************************************
__global__
void arrayProduct(float* a_d, float* b_d, float* c_d, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a_d[i] * b_d[i] = c_d[i];
}

//*******************************************************************
//Method Name: printVals
//Parameters: C
//Purpose: Prints the first and last elements of array C
//*******************************************************************
void printVals(float* C)
{
    printf("C[0] %s", C[0]);
    printf("%c",'\n');
    printf(" C[10239] %s", C[N-1]);
}

int main()
{
    const int TwoBlock = 2;               //used to define 2 blocks
    const int TenBlock = 10;              //used to define 10 blocks
    float A[N] = 0, B[N] = 0, C[N] = 0;   //float arrays inialized to 0
    int size = N * sizeof(float);

    for(int i = 0; i < N; i++)
    {
        A[i] = 2 * i;

        B[i] = 2 * i + 1;
    }

    allocArray(A, B, C, size, TwoBlock); //not cyclic 2 block
    printVals(C);
    for(int i = 0; i < N; i = (i + 1) * 2048) //cyclic 2 block
    {
        allocArray(&A[i], &B[i], &C[i], size/5, TwoBlock);
    }
    printVals(C);
    allocArray(A, B, C, size, TenBlock); //10 block
    printVals(C);

    return 0;
}
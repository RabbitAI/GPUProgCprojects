#include "hip/hip_runtime.h"
using <stdio.h>
using <math.h>

#define Pi 3.14159265

const int N = 8;

int Range = (N/2)-1;

__global__
void ctFFT(float **even_d, float **odd_d, float *real_d, float *imag_d)
{
       int n = threadIdx.x; 
       evenComplex(n, even_d, real_d, imag_d);
       oddComplex(n, odd_d, real_d, imag_d);
}

__device__
void evenComplex(int n; float **even_d, float *real_d, float *imag_d)
{
       real_d[n]= (2*n)*cos(((2*Pi)*(2*n))/N);
       imag[n]= -1*((2*n)*sin((((2*Pi)*(2*n))/N));
       even_d[n][0]= real_d[n];
       even_d[n][1]=imag_d[n];
}

__device__
void oddComplex(int n; float **odd_d, float *real_d, float *imag_d)
{
       real_d[n]= (2*n+1)*cos(((2*Pi)*(2*n+1))/N);
       imag[n]= -1*((2*n+1)*sin((((2*Pi)*(2*n+1))/N));
       odd_d[n][0]= real_d[n];
       odd_d[n][1]=imag_d[n];
}

int main()
{
  float even[Range][2], odd[Range][2], real[Range], imag[Range];
  float **even_d, **odd_d, *real_d, *imag_d;

  for(int i = 0; i < Range; i++)
  {
      even[i] = 2 * i;
      odd[i] = 2 * i + 1;
  }

  hipMalloc((void**)even_d[Range], 2);
  hipMalloc((void**)odd_d[Range], 2);
  hipMalloc((void**)real_d, Range);
  hipMalloc((void**)imag_d, Range);

  hipMemcpy(even_d, even, Range, hipMemcpyHostToDevice);
  hipMemcpy(odd_d, odd, Range, hipMemcpyHostToDevice);

  dim3 dimGrid(1,1);
  dim3 dimBlock(Range, Range);

  ctFFT<<<dimGrid, dimBlock>>>(even_d, odd_d, real, imag);

  return 0;
}